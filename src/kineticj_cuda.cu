#include "hip/hip_runtime.h"
#include "kineticj.cuh"
#include "vecs.cuh"
#include "cusp/complex.h"
#include "constants.hpp"
#include "grid_sizes.hpp"
#include "iostream"
#include "stdio.h"

// used for complex
// usinfg namespace is probably not a good idea here...lazy
using namespace cusp;

// Host wrapper functions
void copyToDevice(complex<float> *j1xc, float *thisT, float *tJp, float *hanningWeight, float *r_kjGrid, C3Vec *e1Re_XYZ_kjGrid,
                  C3Vec *e1Im_XYZ_kjGrid, CParticle_PODS *particles_XYZ_PODS, CParticle_PODS *particles_XYZ_0_PODS,  float *xGrid, C3Vec *b0_CYL_kjGrid,
                  float *df0_dv, complex<float> *all_j1xc, params *p, gpu_mem *gmem)
{
    std::cout <<"copying memory"<<std::endl;

    size_t bytes;

    bytes = sizeof(complex<float>)*p->nXGrid*p->nJp;
    hipMalloc((void**)&(gmem->j1xc),bytes);
    hipMemcpy(gmem->j1xc, j1xc,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*p->nSteps;
    hipMalloc((void**)&(gmem->thisT),bytes);
    hipMemcpy(gmem->thisT, thisT,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*p->nJp;
    hipMalloc((void**)&(gmem->tJp),bytes);
    hipMemcpy(gmem->tJp, tJp,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*p->nSteps;
    hipMalloc((void**)&(gmem->hanningWeight),bytes);
    hipMemcpy(gmem->hanningWeight, hanningWeight,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*_N_DATA;
    hipMalloc((void**)&(gmem->r_kjGrid),bytes);
    hipMemcpy(gmem->r_kjGrid, r_kjGrid,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(C3Vec)*_N_DATA;
    hipMalloc((void**)&(gmem->e1Re_XYZ_kjGrid),bytes);
    hipMemcpy(gmem->e1Re_XYZ_kjGrid, e1Re_XYZ_kjGrid,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(C3Vec)*_N_DATA;
    hipMalloc((void**)&(gmem->e1Im_XYZ_kjGrid),bytes);
    hipMemcpy(gmem->e1Im_XYZ_kjGrid, e1Im_XYZ_kjGrid,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(CParticle_PODS)*p->nV;
    hipMalloc((void**)&(gmem->particles_XYZ_PODS),bytes);
    hipMemcpy(gmem->particles_XYZ_PODS, particles_XYZ_PODS,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(CParticle_PODS)*p->nV;
    hipMalloc((void**)&(gmem->particles_XYZ_0_PODS),bytes);
    hipMemcpy(gmem->particles_XYZ_0_PODS, particles_XYZ_0_PODS,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*p->nXGrid;
    hipMalloc((void**)&(gmem->xGrid),bytes);
    hipMemcpy(gmem->xGrid, xGrid,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(C3Vec)*_N_DATA;
    hipMalloc((void**)&(gmem->b0_CYL_kjGrid),bytes);
    hipMemcpy(gmem->b0_CYL_kjGrid, b0_CYL_kjGrid,  bytes,  hipMemcpyHostToDevice);

    bytes = sizeof(float)*p->nV;
    hipMalloc((void**)&(gmem->df0_dv),bytes);
    hipMemcpy(gmem->df0_dv, df0_dv,  bytes,  hipMemcpyHostToDevice);

    // Create following local arrays
    bytes = sizeof(complex<float>)*p->nV*p->nXGrid;
    hipMalloc((void**)&(gmem->all_j1xc), bytes);
    hipMemcpy(gmem->all_j1xc, all_j1xc,  bytes,  hipMemcpyHostToDevice);

    //bytes = sizeof(C3Vec)*p->nSteps*p->nV*p->nXGrid;
    //hipMalloc((void**)&(gmem->thisOrbitE_re_XYZ), bytes);

    //bytes = sizeof(C3Vec)*p->nSteps*p->nV*p->nXGrid;
    //hipMalloc((void**)&(gmem->thisOrbitE_im_XYZ), bytes);

    //bytes = sizeof(C3Vec)*p->nSteps*p->nV*p->nXGrid;
    //hipMalloc((void**)&(gmem->thisOrbit_XYZ), bytes);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}

void launchKernel(params *p, gpu_mem *gmem)
{

  std::cout <<"kernel"<<std::endl;

  // Blarg
  int block_size = 32;
  int num_blocks = ceilf((p->nXGrid*p->nV)/(float)block_size);
  
  std::cout<<"nxGrid: "<<p->nXGrid<<std::endl;
  std::cout<<"num blocks: "<<num_blocks<<std::endl;

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(low_mem_kernel),hipFuncCachePreferL1);

  low_mem_kernel<<<num_blocks, block_size>>>(gmem->j1xc, gmem->thisT, gmem->tJp, gmem->hanningWeight, gmem->r_kjGrid, gmem->e1Re_XYZ_kjGrid,
                               gmem->e1Im_XYZ_kjGrid, gmem->particles_XYZ_PODS, gmem->particles_XYZ_0_PODS,  gmem->xGrid, gmem->b0_CYL_kjGrid,
                               gmem->df0_dv, gmem->all_j1xc, *p);

    // check for error
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}

void copyToHost(complex<float> *j1xc, complex<float> *all_j1xc, params *p, gpu_mem *gmem)
{

    std::cout <<"copying memory"<<std::endl;

    size_t bytes;
    bytes = sizeof(complex<float>)*p->nXGrid*p->nJp;
    hipMemcpy(j1xc, gmem->j1xc,  bytes,  hipMemcpyDeviceToHost);

    bytes = sizeof(complex<float>)*p->nXGrid*p->nV;
    hipMemcpy(all_j1xc, gmem->all_j1xc,  bytes,  hipMemcpyDeviceToHost);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}

__device__ C3Vec kj_interp1D( float x, const float xVec[], const C3Vec yVec[], int n, int &stat )
{
	float _x, x0, x1;

	if(x<xVec[0]||x>xVec[n-1]||stat>0) {
			// Particle absorbing walls
			++stat;
			return C3Vec(0.0f,0.0f,0.0f);
	}

	_x = (x-xVec[0])/(xVec[n-1]-xVec[0])*(n-1);
	int xF = floorf(_x);
	int xC = ceilf(_x);

	x0 = floorf(_x);
	x1 = ceilf(_x);
	
	// Catch for particle at point
	if(xF==xC) {
		return yVec[xF];
	}
	else {

		C3Vec y0 = yVec[xF];
		C3Vec y1 = yVec[xC];
		C3Vec tmpA = y0+(_x-x0)*(y1-y0)/(x1-x0);
		return y0+(_x-x0)*(y1-y0)/(x1-x0);
	}
}

// Zero-order orbits
__device__ C3Vec rk4_evalf ( CParticle_PODS &p, const float &t,
                const C3Vec &v_XYZ, const C3Vec &x,
                const C3Vec b0_CYL[],
                const float r[],
                const int n ) 
 {

	// Interpolate b0 at location in CYL
	
	float _r = sqrtf ( powf(x.c1,2.0f) + powf(x.c2,2.0f) );
	float _p = atan2f ( x.c2, x.c1 );

	C3Vec thisb0_CYL, thisb0_XYZ;

	thisb0_CYL = kj_interp1D ( _r, r, b0_CYL, n, p.status );

	thisb0_XYZ = C3Vec( cosf(_p)*thisb0_CYL.c1-sinf(_p)*thisb0_CYL.c2,
					sinf(_p)*thisb0_CYL.c1+cosf(_p)*thisb0_CYL.c2,
					thisb0_CYL.c3 );

	C3Vec thisv_x_b0 ( v_XYZ.c2*thisb0_XYZ.c3-v_XYZ.c3*thisb0_XYZ.c2, 
					-1.0f*(v_XYZ.c1*thisb0_XYZ.c3-v_XYZ.c3*thisb0_XYZ.c1), 
					v_XYZ.c1*thisb0_XYZ.c2-v_XYZ.c2*thisb0_XYZ.c1);

	return thisv_x_b0*(p.q/p.m);	
}

// Zero-order orbits
__device__ void rk4_move ( CParticle_PODS &p, const float &dt, const float &t0,
                                const C3Vec b0[], const float r[], const int n) {

		C3Vec k1, k2, k3, k4, yn1, x1, x2, x3, x4, xn1; 

		C3Vec yn0(p.v_c1,p.v_c2,p.v_c3), xn0(p.c1, p.c2, p.c3);
		k1 = rk4_evalf ( p, t0 + 0.0f*dt, yn0         , xn0         , b0, r, n ) * dt;	
		x1 = yn0 * dt;
		k2 = rk4_evalf ( p, t0 + 0.5f*dt, yn0 + 0.5f*k1, xn0 + 0.5f*x1, b0, r, n ) * dt;	
		x2 = (yn0 + 0.5f*k1) * dt;
		k3 = rk4_evalf ( p, t0 + 0.5f*dt, yn0 + 0.5f*k2, xn0 + 0.5f*x2, b0, r, n ) * dt;	
		x3 = (yn0 + 0.5f*k2) * dt;
		k4 = rk4_evalf ( p, t0 + 1.0f*dt, yn0 + 1.0f*k3, xn0 + 1.0f*x3, b0, r, n ) * dt;	
		x4 = (yn0 + 1.0f*k3) * dt;

		//printf("dt: %f\n",dt);
		//printf("k1: %f, k2: %f, k3: %f, k4: %f\n", k1.c1, k2.c1, k3.c1, k4.c1);

		yn1 = yn0 + 1.0f/6.0f * (k1+2.0f*k2+2.0f*k3+k4);
		xn1 = xn0 + 1.0f/6.0f * (x1+2.0f*x2+2.0f*x3+x4);

		p.c1 = xn1.c1;
		p.c2 = xn1.c2;
		p.c3 = xn1.c3;
		p.v_c1 = yn1.c1;
		p.v_c2 = yn1.c2;
		p.v_c3 = yn1.c3;
}

// Need to find correct memory space for these. Constant cache, shared, etc...
__global__ void low_mem_kernel(complex<float> *j1xc, float *thisT, float *tJp, float *hanningWeight, float *r_kjGrid, C3Vec *e1Re_XYZ_kjGrid,
			       C3Vec *e1Im_XYZ_kjGrid, CParticle_PODS *particles_XYZ_PODS, CParticle_PODS *particles_XYZ_0_PODS,  float *xGrid, C3Vec *b0_CYL_kjGrid, 
			       float *df0_dv, complex<float> *all_j1xc, const __restrict params p)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    //int nXGrid = p.nXGrid;

    float dv = p.dv;
    int nSteps = p.nSteps;
    int nV = p.nV;
    double dtMin = p.dtMin;
    double wrf = p.wrf;
    //int nJp = p.nJp;

	int iX = tid / nV;
	int iP = tid % nV;

    //if(iX < nXGrid ) {

	//printf("tid: %i, iX: %i, iP: %i\n", tid, iX, iP);
	//printf("dv: %f, nSteps: %i, nV: %i, dtMin: %e, wrf: %f, nJp: %i\n", dv, nSteps, nV, dtMin, wrf, nJp);

    int i, istat;
    complex<float> this_j1xc;
    complex<float> f1c;

    CParticle_PODS thisParticle_XYZ;

    //for(iP=0;iP<nV;iP++) {

        thisParticle_XYZ = particles_XYZ_PODS[iP];
        thisParticle_XYZ.c1 = xGrid[iX];

        double qOverm =  thisParticle_XYZ.q/thisParticle_XYZ.m;
        float qe = thisParticle_XYZ.q;
        float h = dv * qe;
        
        // generate orbit and get time-harmonic e along it
        C3Vec e1ReTmp_XYZ, e1ImTmp_XYZ;
       
        // get Jp(t) for this spatial point
        C3VecI thisEc;
        C3VecI thisV1c;
       
        for(i=0;i<nSteps;i++) {

			e1ReTmp_XYZ = C3Vec(0,0,0);
 			e1ImTmp_XYZ = C3Vec(0,0,0);

            if(thisParticle_XYZ.status==0) {
                
                rk4_move ( thisParticle_XYZ, dtMin, thisT[i], b0_CYL_kjGrid, r_kjGrid, _N_DATA );
   
                if(thisParticle_XYZ.status==0) {
		    		istat = 0;
                    e1ReTmp_XYZ = kj_interp1D ( thisParticle_XYZ.c1, r_kjGrid, e1Re_XYZ_kjGrid, _N_DATA, istat);
		    		istat = 0;
                    e1ImTmp_XYZ = kj_interp1D ( thisParticle_XYZ.c1, r_kjGrid, e1Im_XYZ_kjGrid, _N_DATA, istat);
                }
            }           

            float tTmp = thisT[i];
            float weight = hanningWeight[i];
            float phs = -(wrf*tTmp);
            
            thisEc = C3VecI(
                            weight*complex<float>(
                                                  e1ReTmp_XYZ.c1*cosf(phs)-e1ImTmp_XYZ.c1*sinf(phs),
                                                  e1ImTmp_XYZ.c1*cosf(phs)+e1ReTmp_XYZ.c1*sinf(phs)),
                            weight*complex<float>(
                                                  e1ReTmp_XYZ.c2*cosf(phs)-e1ImTmp_XYZ.c2*sinf(phs),
                                                  e1ImTmp_XYZ.c2*cosf(phs)+e1ReTmp_XYZ.c2*sinf(phs)),
                            weight*complex<float>( 
                                                  e1ReTmp_XYZ.c3*cosf(phs)-e1ImTmp_XYZ.c3*sinf(phs),
                                                  e1ImTmp_XYZ.c3*cosf(phs)+e1ReTmp_XYZ.c3*sinf(phs))
                            );
            
            int N = nSteps - 1;
            float A = i % N;
            float B = A / N;
            int factor = ceilf(B)+1;
            
            thisV1c += -qOverm * dtMin/2 * ( factor * thisEc);
        }
      
        f1c = -thisV1c.c1*df0_dv[iP];
        
        float v0_i = particles_XYZ_0_PODS[iP].v_c1;
        
        int N = nV - 1;
        float A = iP % N;
        float B = A / N;
        int factor = ceilf(B)+1;
        
        //this_j1xc += h/2 * ( factor * v0_i*f1c);
		all_j1xc[iX*nV+iP] = h/2 * ( factor * v0_i*f1c);
    //}
    
    //[iX][0]
    //j1xc[iX*nJp] = this_j1xc;

    //}
}
