#include "hip/hip_runtime.h"
#include "rotation.hpp"
#include "constants.hpp"

void transpose(float A[][3])
{

    float B[3][3];

    B[0][0] = A[0][0];
    B[1][0] = A[0][1];
    B[2][0] = A[0][2];

    B[0][1] = A[1][0];
    B[1][1] = A[1][1];
    B[2][1] = A[1][2];

    B[0][2] = A[2][0];
    B[1][2] = A[2][1];
    B[2][2] = A[2][2];

    A = B;
}

C3<float> rot_XYZ_to_abp(const C3<float> A_XYZ, const C3<float> bUnit_XYZ, const int direction)
{

    // If direction<1 then the inverse rotation is applied, i.e., abp_to_XYZ

    C3<float> A_abp;

    C3<float> xu_xyz(1, 0, 0);
    C3<float> yu_xyz(0, 1, 0);
    C3<float> zu_xyz(0, 0, 1);

    C3<float> pu_xyz = bUnit_XYZ;

    // alp is mostly in the +/- x / r direction depending on B toroidal direction
    // bet is mostly z direction

    C3<float> a_xyz = cross(zu_xyz, pu_xyz);
    C3<float> au_xyz = a_xyz / mag(a_xyz);
    //C3<float> au_xyz = ( static_cast<float>(1.0) / mag(a_xyz) ) * a_xyz;

    C3<float> b_xyz = cross(pu_xyz, au_xyz);
    C3<float> bu_xyz = b_xyz / mag(b_xyz);

#if DEBUG_ROTATION >= 1
    
    std::cout<< "bUnit_XYZ: "<<bUnit_XYZ<<std::endl;
    std::cout<< "a_xyz: "<<a_xyz<<std::endl;
    std::cout<< "mag(a_xyz): " << mag(a_xyz) << std::endl;

    C3<float> au_xyz2 = au_xyz;
    C3<float> bu_xyz2 = bu_xyz;
    C3<float> pu_xyz2 = pu_xyz;

    std::cout << "au_xyz: " << au_xyz << std::endl;
    std::cout << "bu_xyz: " << bu_xyz << std::endl;
    std::cout << "pu_xyz: " << pu_xyz << std::endl;
#endif

    // Rotation 1

    float theta = acos(dot(xu_xyz, au_xyz));

#if DEBUG_ROTATION >= 1
    std::cout << "xu_xyz: " << xu_xyz << std::endl;
    std::cout << "au_xyz: " << au_xyz << std::endl;
    std::cout << "dot: " << dot(xu_xyz, au_xyz) << std::endl;

    std::cout << "theta [rad]: " << theta << std::endl;
    std::cout << "theta [deg]: " << theta * 180.0 / physConstants::pi << std::endl;
#endif

    float q0 = cos(theta / 2.0);
    float q1 = sin(theta / 2.0) * (-zu_xyz.c1);
    float q2 = sin(theta / 2.0) * (-zu_xyz.c2);
    float q3 = sin(theta / 2.0) * (-zu_xyz.c3);

    // Construct the rotation matrix

    float rot1[3][3];

    rot1[0][0] = pow(q0, 2.0) + pow(q1, 2.0) - pow(q2, 2.0) - pow(q3, 2.0);
    rot1[0][1] = 2 * (q1 * q2 - q0 * q3);
    rot1[0][2] = 2 * (q1 * q3 + q0 * q2);
    rot1[1][0] = 2 * (q2 * q1 + q0 * q3);
    rot1[1][1] = pow(q0, 2.0) - pow(q1, 2.0) + pow(q2, 2.0) - pow(q3, 2.0);
    rot1[1][2] = 2 * (q2 * q3 - q0 * q1);
    rot1[2][0] = 2 * (q3 * q1 - q0 * q2);
    rot1[2][1] = 2 * (q3 * q2 + q0 * q1);
    rot1[2][2] = pow(q0, 2.0) - pow(q1, 2.0) - pow(q2, 2.0) + pow(q3, 2.0);

    if (direction < 0) {
        transpose(rot1);
    }

    au_xyz = rot1 * au_xyz;
    bu_xyz = rot1 * bu_xyz;
    pu_xyz = rot1 * pu_xyz;

#if DEBUG_ROTATION >= 1
    std::cout << "au_rtz 1: " << au_xyz << std::endl;
    std::cout << "bu_rtz 1: " << bu_xyz << std::endl;
    std::cout << "pu_rtz 1: " << pu_xyz << std::endl;
#endif

    // Rotation 2

    theta = acos(dot(zu_xyz, pu_xyz));

#if DEBUG_ROTATION >= 1
    std::cout << "theta: " << theta * 180.0 / physConstants::pi << std::endl;
#endif
    q0 = cos(theta / 2.0);
    q1 = sin(theta / 2.0) * (-xu_xyz.c1);
    q2 = sin(theta / 2.0) * (-xu_xyz.c2);
    q3 = sin(theta / 2.0) * (-xu_xyz.c3);

    // Construct the rotation matrix

    float rot2[3][3];

    rot2[0][0] = pow(q0, 2.0) + pow(q1, 2.0) - pow(q2, 2.0) - pow(q3, 2.0);
    rot2[0][1] = 2 * (q1 * q2 - q0 * q3);
    rot2[0][2] = 2 * (q1 * q3 + q0 * q2);
    rot2[1][0] = 2 * (q2 * q1 + q0 * q3);
    rot2[1][1] = pow(q0, 2.0) - pow(q1, 2.0) + pow(q2, 2.0) - pow(q3, 2.0);
    rot2[1][2] = 2 * (q2 * q3 - q0 * q1);
    rot2[2][0] = 2 * (q3 * q1 - q0 * q2);
    rot2[2][1] = 2 * (q3 * q2 + q0 * q1);
    rot2[2][2] = pow(q0, 2.0) - pow(q1, 2.0) - pow(q2, 2.0) + pow(q3, 2.0);

    if (direction < 0) {
        transpose(rot2);
    }

    au_xyz = rot2 * au_xyz;
    bu_xyz = rot2 * bu_xyz;
    pu_xyz = rot2 * pu_xyz;

#if DEBUG_ROTATION >= 1
    std::cout << "au_xyz 2: " << au_xyz << std::endl;
    std::cout << "bu_xyz 2: " << bu_xyz << std::endl;
    std::cout << "pu_xyz 2: " << pu_xyz << std::endl;
#endif

    A_abp = rot2 * (rot1 * A_XYZ);

    return A_abp;
}

C3<float> rot_axis_angle(const C3<float> v, const C3<float> u, const float th_deg) {

        // See https://en.wikipedia.org/wiki/Rotation_matrix#In_three_dimensions

        C3<float> result;
        float th = th_deg * physConstants::pi / 180.0;
        float cosTh = cos(th);
        float sinTh = sin(th);

        float ux = u.c1;
        float uy = u.c2;
        float uz = u.c3;

        float R11 = cosTh + std::pow(ux,2)*(1-cosTh);
        float R12 = ux*uy*(1-cosTh)-uz*sinTh;
        float R13 = ux*uz*(1-cosTh)+uy*sinTh;

        float R21 = uy*ux*(1-cosTh)+uz*sinTh;
        float R22 = cosTh+std::pow(uy,2)*(1-cosTh);
        float R23 = uy*uz*(1-cosTh)-ux*sinTh;

        float R31 = uz*ux*(1-cosTh)-uy*sinTh;
        float R32 = uz*uy*(1-cosTh)+ux*sinTh;
        float R33 = cosTh+std::pow(uz,2)*(1-cosTh);

        result.c1 = R11 * v.c1 + R12 * v.c2 + R13 * v.c3;
        result.c2 = R21 * v.c1 + R22 * v.c2 + R23 * v.c3;
        result.c3 = R31 * v.c1 + R32 * v.c2 + R33 * v.c3;

        return result;
}
