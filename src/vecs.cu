#include "hip/hip_runtime.h"
#include "constants.hpp"
#include "grid_sizes.hpp"
#include "cusp/complex.h"
#include "vecs.cuh"
#include <vector>

// used for complex
using cusp::complex;
using std::vector;

__host__ __device__ C3Vec::C3Vec () {
    c1=0;
    c2=0;
    c3=0;
}

__host__ __device__ C3Vec::C3Vec ( float _c1, float _c2, float _c3 )
{
    c1=_c1;
    c2=_c2;
    c3=_c3;
}

__host__ __device__ C3Vec::C3Vec ( int _arg ) 
{
    c1=_arg;
    c2=_arg;
    c3=_arg;
}

__host__ __device__ C3VecI::C3VecI ( complex<float> _c1, complex<float> _c2, complex<float> _c3 ) 
{
    c1=_c1;
    c2=_c2;
    c3=_c3;
}

__host__ __device__ C3VecI::C3VecI ()
{
    c1=complex<float>(0.0f,0.0f);
    c2=complex<float>(0.0f,0.0f);
    c3=complex<float>(0.0f,0.0f);
}

__host__ __device__
C3Vec& C3Vec::operator= (const C3Vec &rhs ) {
		if (this != &rhs) {
				c1 = rhs.c1;
				c2 = rhs.c2;
				c3 = rhs.c3;
		}
		return *this;
}

__host__ __device__ 
C3Vec& C3Vec::operator= (const float &rhs){
	c1 = rhs;
	c2 = rhs;
	c3 = rhs;
	return *this;
}

__host__ __device__
C3VecI& C3VecI::operator= (const C3VecI &rhs ) {
		if (this != &rhs) {
				c1 = rhs.c1;
				c2 = rhs.c2;
				c3 = rhs.c3;
		}
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator+= (const C3Vec &rhs ) {
		c1 = c1 + rhs.c1;
		c2 = c2 + rhs.c2;
		c3 = c3 + rhs.c3;
		return *this;
}

__host__ __device__
C3Vec& C3Vec::operator+= (const float &rhs ) {
		c1 = c1 + rhs;
		c2 = c2 + rhs;
		c3 = c3 + rhs;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator-= (const C3Vec &rhs ) {
		c1 = c1 - rhs.c1;
		c2 = c2 - rhs.c2;
		c3 = c3 - rhs.c3;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator-= (const float &rhs ) {
		c1 = c1 - rhs;
		c2 = c2 - rhs;
		c3 = c3 - rhs;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator-= (const C3VecI &rhs ) {
		c1 = c1 - rhs.c1;
		c2 = c2 - rhs.c2;
		c3 = c3 - rhs.c3;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator-= (const float &rhs ) {
		c1 = c1 - rhs;
		c2 = c2 - rhs;
		c3 = c3 - rhs;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator*= (const C3Vec &rhs ) {
		c1 *= rhs.c1;
		c2 *= rhs.c2;
		c3 *= rhs.c3;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator*= (const float &rhs ) {
		c1 *= rhs;
		c2 *= rhs;
		c3 *= rhs;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator/= (const C3Vec &rhs ) {
		c1 /= rhs.c1;
		c2 /= rhs.c2;
		c3 /= rhs.c3;
		return *this;
}
__host__ __device__
C3Vec& C3Vec::operator/= (const float &rhs ) {
		c1 /= rhs;
		c2 /= rhs;
		c3 /= rhs;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator/= (const C3VecI &rhs ) {
		c1 /= rhs.c1;
		c2 /= rhs.c2;
		c3 /= rhs.c3;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator/= (const float &rhs ) {
		c1 /= rhs;
		c2 /= rhs;
		c3 /= rhs;
		return *this;
}
__host__ __device__
C3Vec C3Vec::operator+ (const C3Vec &other) {
		return C3Vec(this->c1+other.c1,this->c2+other.c2,this->c3+other.c3);
}
__host__ __device__
C3Vec C3Vec::operator+ (const float &other) {
		return C3Vec(*this)+=other;
}
__host__ __device__
C3Vec C3Vec::operator- (const C3Vec &other) {
		return C3Vec(*this)-=other;
}
__host__ __device__
C3Vec C3Vec::operator- (const float &other) {
		return C3Vec(*this)-=other;
}
__host__ __device__
C3VecI C3VecI::operator- (const C3VecI &other) {
		return C3VecI(*this)-=other;
}
__host__ __device__
C3VecI C3VecI::operator- (const float &other) {
		return C3VecI(*this)-=other;
}
__host__ __device__
C3Vec C3Vec::operator* (const C3Vec &other) {
		return C3Vec(*this)*=other;
}
__host__ __device__
C3Vec C3Vec::operator* (const float &other) {
		return C3Vec(*this)*=other;
}
__host__ __device__
C3Vec C3Vec::operator/ (const C3Vec &other) {
		return C3Vec(*this)/=other;
}
__host__ __device__
C3Vec C3Vec::operator/ (const float &other) {
		return C3Vec(*this)/=other;
}
__host__ __device__
C3VecI C3VecI::operator/ (const C3VecI &other) {
		return C3VecI(*this)/=other;
}
__host__ __device__
C3VecI C3VecI::operator/ (const float &other) {
		return C3VecI(*this)/=other;
}
// C3VecI 
__host__ __device__
C3VecI& C3VecI::operator+= (const C3VecI &rhs ) {
		c1 += rhs.c1;
		c2 += rhs.c2;
		c3 += rhs.c3;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator+= (const float &rhs ) {
		c1 += rhs;
		c2 += rhs;
		c3 += rhs;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator*= (const C3VecI &rhs ) {
		c1 *= rhs.c1;
		c2 *= rhs.c2;
		c3 *= rhs.c3;
		return *this;
}
__host__ __device__
C3VecI& C3VecI::operator*= (const float &rhs ) {
		c1 *= rhs;
		c2 *= rhs;
		c3 *= rhs;
		return *this;
}
__host__ __device__
C3VecI C3VecI::operator+ (const C3VecI &other) {
		return C3VecI(this->c1+other.c1,this->c2+other.c2,this->c3+other.c3);
}
__host__ __device__
C3VecI C3VecI::operator+ (const float &other) {
		return C3VecI(*this)+=other;
}
__host__ __device__
C3VecI C3VecI::operator* (const C3VecI &other) {
		return C3VecI(*this)*=other;
}
__host__ __device__
C3VecI C3VecI::operator* (const float &other) {
		return C3VecI(*this)*=other;
}

// Global (not member) functions for lhs operators
__host__ __device__
C3Vec operator* ( const float &other, const C3Vec &rhs ) {
		return C3Vec(rhs)*=other;
}
__host__ __device__
C3VecI operator* ( const float &other, const C3VecI &rhs ) {
		return C3VecI(rhs)*=other;
}
__host__ __device__
C3Vec operator+ ( const C3Vec &other, const C3Vec &rhs) {
		return C3Vec(other.c1+rhs.c1,other.c2+rhs.c2,other.c3+rhs.c3);
}
__host__ __device__
C3VecI operator+ ( const C3VecI &other, const C3VecI &rhs) {
		return C3VecI(other.c1+rhs.c1,other.c2+rhs.c2,other.c3+rhs.c3);
}
__host__ __device__
C3Vec pow ( const C3Vec &in, const int arg ) {
		C3Vec out;
		out.c1 = pow(in.c1,arg);
		out.c2 = pow(in.c2,arg);
		out.c3 = pow(in.c3,arg);
		return out;
}
__host__ __device__
C3Vec sqrt ( const C3Vec &in ) {
		C3Vec out;
		out.c1 = sqrt(in.c1);
		out.c2 = sqrt(in.c2);
		out.c3 = sqrt(in.c3);
		return out;
}
__host__ __device__
C3Vec atan2 ( const C3Vec &Y, const C3Vec &X ) {
		C3Vec out;
		out.c1 = atan2(Y.c1,X.c1);
		out.c2 = atan2(Y.c2,X.c2);
		out.c3 = atan2(Y.c3,X.c3);
		return out;
}

/*
__host__
float maxC3VecAbs ( const vector<C3Vec> &input ) {

        vector<float> inputAbs(input.size());
        for(int i=0;i<input.size();i++) {
                inputAbs[i] = sqrt(pow(input[i].c1,2)+pow(input[i].c2,2)+pow(input[i].c3,2));
        }
        return *max_element(inputAbs.begin(),inputAbs.end());
}
__host__
C3Vec intC3VecArray ( const vector<float> &x, const vector<C3Vec> &f ) {

        C3Vec result;
        float h = x[1]-x[0];
        for(int i=1;i<f.size();i++) {
                result += h/2.0*(f[i-1]+f[i]);
        }

        return result;
}
__host__
C3VecI intC3VecArray ( const vector<float> &x, const vector<C3VecI> &f ) {

        C3VecI result;
        float h = x[1]-x[0];
        for(int i=1;i<f.size();i++) {
                result += h/2.0*(f[i-1]+f[i]);
        }

        return result;
}
__host__
C3VecI intC3VecArray ( const float x[], const vector<C3VecI> &f ) {

        C3VecI result;
        float h = x[1]-x[0];
        for(int i=1;i<f.size();i++) {
                result += h/2.0*(f[i-1]+f[i]);
        }

        return result;
}
__host__
C3Vec intC3VecArray ( const float x[], const vector<C3Vec> &f ) {

        C3Vec result;
        float h = x[1]-x[0];
        for(int i=1;i<f.size();i++) {
                result += h/2.0*(f[i-1]+f[i]);
        }

        return result;
}
*/
