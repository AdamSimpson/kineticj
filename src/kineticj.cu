#include "hip/hip_runtime.h"
#include "c3vec.hpp"
#ifdef __HIPCC__
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#endif


#include "constants.hpp"
#include "cparticle.hpp"
#include "createParticles.hpp"
#include "cspecies.hpp"
#include "interp.hpp"
#include "read_e_field.hpp"
#include "read_gc_file.hpp"
#include "rk4.hpp"
#include "rotation.hpp"
#include <algorithm>
#include <assert.h>
#include <complex>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <libconfig.h++>
#include <netcdf>
#include <new> // for stl::bad_alloc
#include <omp.h>
#include <string>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <vector>
#include <numeric>

#if CLOCK >= 1
#include <ctime>
#endif

#if USEPAPI >= 1
#include <papi.h>
#endif

#if LOWMEM_USEPAPI >= 1
#include <papi.h>
#endif

//#include <google/profiler.h>

#ifdef __CUDA_ARCH__
#define PRINT cuPrintf
#else
#define PRINT printf
#endif

using namespace std;
using namespace netCDF;
using namespace exceptions;

// Calculate the jP given some know E and f(v)

int main(int argc, char** argv)
{

    // Make sure the "output/" directory exists

    stringstream outputDirName;
    outputDirName << "output/";

    // check directory exists
    struct stat st;
    int dirTest = stat(outputDirName.str().c_str(), &st);
    if (dirTest != 0) {
        cout << "Had to create output/ directory" << endl;
        int mkDirStat = mkdir(outputDirName.str().c_str(),
            S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    }

#if CLOCK >= 1
    clock_t ProgramTime = clock();
#endif

#if (USEPAPI >= 1 || LOWMEM_USEPAPI >= 1)
    float realTime0, cpuTime0, realTime = 0, cpuTime = 0, mFlops = 0;
    long long flpIns0, flpIns = 0;
    int papiReturn;

    cpuTime0 = cpuTime;
    realTime0 = realTime;
    flpIns0 = flpIns;
    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    if (papiReturn < 0) {
        cout << "ERROR: PAPI Failed to initialize with error code: " << papiReturn
             << endl;
        cout << "ERROR: See papi.h for error code explanations " << endl;
        exit(1);
    }
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime, cpuTime, flpIns, mFlops);

    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    if (papiReturn < 0) {
        cout << "ERROR: PAPI Failed to initialize with error code: " << papiReturn
             << endl;
        cout << "ERROR: See papi.h for error code explanations " << endl;
        exit(1);
    } else {
        cout << "PAPI called successfully with return code: " << papiReturn << endl;
    }
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime, cpuTime, flpIns, mFlops);
#endif

    // Read config file

    libconfig::Config cfg;
    string cfgName = "kj.cfg";

    try {
        cfg.readFile(cfgName.c_str());
    } catch (const libconfig::FileIOException& fioex) {
        std::cerr << "I/O error while reading file." << std::endl;
        return (EXIT_FAILURE);
    } catch (const libconfig::ParseException& pex) {
        std::cerr << "Parse error at " << pex.getFile() << ":" << pex.getLine()
                  << " - " << pex.getError() << std::endl;
        return (EXIT_FAILURE);
    }

    int species_number = cfg.lookup("species_number");

    // Read E
    string eField_fName = cfg.lookup("eField_fName");
    vector<C3Vec> e1Re_CYL, e1Im_CYL, b1Re_CYL, b1Im_CYL;
    vector<C3VecI> e1_CYL, b1_CYL;
    vector<C3Vec> b0_CYL, b0_XYZ;
    vector<float> r, n_m3;
    float freq;
    int eReadStat = read_e_field(eField_fName, species_number, freq, r, n_m3, e1_CYL, b1_CYL,
        e1Re_CYL, e1Im_CYL, b1Re_CYL, b1Im_CYL, b0_CYL);

    // Read GC terms
    string gc_fName = cfg.lookup("gc_fName");
    vector<C3Vec> curv_CYL, grad_CYL;
    std::vector<float> r_gc, bDotGradB;
    int gcReadStat = read_gc_file(gc_fName, r_gc, curv_CYL, grad_CYL, bDotGradB);

    float wrf = freq * 2 * physConstants::pi;
    float xGridMin = cfg.lookup("xGridMin");
    float xGridMax = cfg.lookup("xGridMax");
    int nXGrid = cfg.lookup("nXGrid");
    cout << "nXGrid: " << nXGrid << endl;

    vector<float> xGrid(nXGrid);
    vector<float> density_m3(nXGrid);
    vector<float> T_keV(nXGrid);
    vector<float> wrf_wc(nXGrid);
    vector<float> bMag_kjGrid(nXGrid);

    float xGridRng = 0;
    float xGridStep = 0;

    if (nXGrid > 1) {
        xGridRng = xGridMax - xGridMin;
        xGridStep = xGridRng / (nXGrid - 1);
    }

    for (int iX = 0; iX < nXGrid; iX++) {
        xGrid[iX] = xGridMin + iX * xGridStep;
        int iStat;
        density_m3[iX] = kj_interp1D(xGrid[iX], r, n_m3, iStat);
        C3Vec this_b0 = kj_interp1D(xGrid[iX], r, b0_CYL, iStat);
        bMag_kjGrid[iX] = mag(this_b0);
        T_keV[iX] = 2.0; // kj_interp1D(xGrid[iX],r,n_m3);
    }

    float MaxB0 = *max_element(bMag_kjGrid.begin(), bMag_kjGrid.end());

#if USEPAPI >= 1
    cpuTime0 = cpuTime;
    realTime0 = realTime;
    flpIns0 = flpIns;
    papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
    printf("\nStartup performance:\n");
    printf("Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
        realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif

    float nRFCycles = cfg.lookup("nRFCycles");
    float nStepsPerCycle = cfg.lookup("nStepsPerCycle");
    float tRF = (2 * physConstants::pi) / wrf;
    int nJpCycles = cfg.lookup("nJpCycles");
    int nJpPerCycle = cfg.lookup("nJpPerCycle");
    int nPhi = cfg.lookup("nPhi");
    int nJp = nJpCycles * nJpPerCycle;
    float dtJp = tRF / nJpPerCycle;
    int istat = 0;
    int nPx = cfg.lookup("nPx");
    int nPy = cfg.lookup("nPy");
    int nPz = cfg.lookup("nPz");
    float amu = cfg.lookup("species_amu");
    float Z = cfg.lookup("species_Z");
    int nThermal = cfg.lookup("nThermal");
    long int nP = nPx * nPy * nPz;
    float wc = Z * physConstants::e * MaxB0 / (amu * physConstants::mi);
    float cyclotronPeriod = 2 * physConstants::pi / wc;
    float dtMin = -cyclotronPeriod / nStepsPerCycle;
    int nSteps = nRFCycles * tRF / abs(dtMin) + 1;

    for (int iX = 0; iX < nXGrid; iX++) {
        float this_wc = Z * physConstants::e * bMag_kjGrid[iX] / (amu * physConstants::mi);
        wrf_wc[iX] = wrf / this_wc;
    }

#if PRINT_INFO >= 1
    cout << "dtMin [s]: " << dtMin << endl;
    cout << "Cyclotron Period: " << cyclotronPeriod << endl;
    cout << "RF Period: " << tRF << endl;
    cout << "nSteps: " << nSteps << endl;
    cout << "nStepsPerCycle: " << nStepsPerCycle << endl;
    cout << "freq: " << freq << endl;
    cout << "Max B0: " << MaxB0 << endl;
#endif

    vector<float> thisT;
    try {
        thisT.resize(nSteps);
    } catch (const std::bad_alloc& error) {
        cout << "Allocation error at " << __FILE__ << __LINE__ << endl;
        cout << error.what();
    }

    for (int i = 0; i < nSteps; i++) {
        thisT[i] = i * dtMin; //+1.5*dtMin;
    }

    vector<float> hanningWeight(nSteps);
    vector<float> expWeight(nSteps);
    vector<float> linearWeight(nSteps);
    for (int i = 0; i < nSteps; i++) {
        // linearWeight[i]=thisT[i]*1.0/(tRF*nRFCycles)+1.0;
        hanningWeight[i] = 0.5 * (1 - cos(2 * physConstants::pi * i / (nSteps - 1))); // Regular
        // hanningWeight[i]=0.5*(1-cos(2*physConstants::pi*i/(nSteps*0.25-1))); //Sharper
        // hanningWeight[i] = linearWeight[i];
        if (i < nSteps / 2)
            hanningWeight[i] = 1; // Regular
        // if(i<nSteps*7.0/8.0) hanningWeight[i]=1; //Sharper
        // complex<float> _i (0.0,1.0);
        // complex<float> wrf_c (wrf,wrf*0.0025);
        // expWeight[i] = 1.0;//abs(exp(-_i*wrf_c*thisT[i]));
        // hanningWeight[i] = hanningWeight[i] * expWeight[i];
    }

    vector<vector<float> > j1x(nXGrid), j1y(nXGrid), j1z(nXGrid);
    vector<complex<float> > j1xc(nXGrid), j1yc(nXGrid), j1zc(nXGrid);

#if defined(_OPENMP)
    int nThreads, tid, spoken = 0;
#endif

#if CLOCK >= 1
        clock_t startTimeFunctor = clock();
#endif

    float dv;

    // Create worklist of nX * nP particles

    long int nWork = nXGrid * nP;

    vector<CParticle> particleWorkList;
    for (int iX = 0; iX < nXGrid; iX++) {

        vector<CParticle> moreWork(
            create_particles(xGrid[iX], amu, Z, T_keV[iX], density_m3[iX], nPx, nPy,
                nPz, nThermal, dv, r, b0_CYL));

        particleWorkList.insert( particleWorkList.end(), moreWork.begin(), moreWork.end() );
    }

#ifdef __HIPCC__
    //thrust::device_vector<CParticle> particleWorkList_device = particleWorkList;
#endif

    // Create the vx,vy,vz iterators

    vector<float> vx(nWork,0);
    vector<float> vy(nWork,0);
    vector<float> vz(nWork,0);

    transform( vx.begin(), vx.end(), particleWorkList.begin(), vx.begin(), set_vx() );
    transform( vy.begin(), vy.end(), particleWorkList.begin(), vy.begin(), set_vy() );
    transform( vz.begin(), vz.end(), particleWorkList.begin(), vz.begin(), set_vz() );

    // Move particles
    cout << "Moving particles with for_each ..." << endl;

    // Velocity space calculation

    vector<C3Vec> df0_dv_XYZ(nWork,0);
    vector<C3VecI> E1(nWork,0);
    vector<C3VecI> B1(nWork,0);
    vector<C3VecI> vCrossB(nWork,0);
    vector<C3VecI> vCrossB_E1(nWork,0);
    vector<complex<float> > forceDotGradf0(nWork,0);
    vector<complex<float> > dtIntegral(nWork,0);
    vector<complex<float> > f1(nWork,0);
    vector<complex<float> > vxf1(nWork,0);
    vector<complex<float> > vyf1(nWork,0);
    vector<complex<float> > vzf1(nWork,0);

    for (int i = 0; i < nSteps; i++) {

        float dtIntFac = 1;
        if (i > 0) dtIntFac = 2;

        dtIntFac = dtMin / 2.0 * dtIntFac;

        // Move particle
        for_each( particleWorkList.begin(), particleWorkList.end(), moveParticle(dtMin, r, b0_CYL) ); 
       
        // df0(v)/dv 
        transform( particleWorkList.begin(), particleWorkList.end(), df0_dv_XYZ.begin(), get_df0_dv() ); 

        // E1(x) 
        transform( particleWorkList.begin(), particleWorkList.end(), E1.begin(), getPerturbedField(r,e1_CYL,nPhi,hanningWeight[i]) ); 

        // B1(x) 
        transform( particleWorkList.begin(), particleWorkList.end(), B1.begin(), getPerturbedField(r,b1_CYL,nPhi,hanningWeight[i]) ); 

        // v x B1 
        transform( particleWorkList.begin(), particleWorkList.end(), B1.begin(), vCrossB.begin(), vCross() );

        // E1 + v x B1
        transform( E1.begin(), E1.end(), vCrossB.begin(), vCrossB_E1.begin(), std::plus<C3VecI>() );

        //  (E1 + v x B1) . grad_v(f0(v))
        transform( vCrossB_E1.begin(), vCrossB_E1.end(), df0_dv_XYZ.begin(), forceDotGradf0.begin(), doDotProduct() );

        // int( (E1 + v x B1) . grad_v(f0(v)), dt ) via running dt integral
        transform( dtIntegral.begin(), dtIntegral.end(), forceDotGradf0.begin(), dtIntegral.begin(), runningIntegral(dtIntFac) );

        // f1(v) = -q/m * int( (E1 + v x B1) . grad_v(f0(v)), dt )
        transform( dtIntegral.begin(), dtIntegral.end(), particleWorkList.begin(), f1.begin(), multiplyByChargeOverMass() ); 

        // q . f1(v) // first step in velocity momemnt for current calculation 
        transform( f1.begin(), f1.end(), particleWorkList.begin(), f1.begin(), multiplyByCharge() ); 

        // q . vx . f1(v) 
        transform( f1.begin(), f1.end(), vx.begin(), vxf1.begin(), std::multiplies< complex<float> >() ); 

        // q . vy . f1(v) 
        transform( f1.begin(), f1.end(), vy.begin(), vyf1.begin(), std::multiplies< complex<float> >() ); 

        // q . vz . f1(v) 
        transform( f1.begin(), f1.end(), vz.begin(), vzf1.begin(), std::multiplies< complex<float> >() ); 
    }

    // Reduce velocity space to current via the first velocity moment

    for (int i=0;i<nXGrid;i++) {
        j1xc[i] = dv * accumulate( vxf1.begin()+nP*i, vxf1.begin()+nP*i+nP, complex<float>(0) );
        j1yc[i] = dv * accumulate( vyf1.begin()+nP*i, vyf1.begin()+nP*i+nP, complex<float>(0) );
        j1zc[i] = dv * accumulate( vzf1.begin()+nP*i, vzf1.begin()+nP*i+nP, complex<float>(0) );
        cout << j1xc[i].real() << "  " << j1xc[i].imag() << endl;
    }

    stringstream ncjPFileName2("jP2.nc");

    NcFile ncjPFile(ncjPFileName2.str().c_str(), NcFile::replace);

    NcDim nc_nX = ncjPFile.addDim("nJp", nXGrid);

    NcVar nc_x = ncjPFile.addVar("x", ncFloat, nc_nX);

    NcVar nc_j1xc_re = ncjPFile.addVar("j1xc_re", ncFloat, nc_nX);
    NcVar nc_j1xc_im = ncjPFile.addVar("j1xc_im", ncFloat, nc_nX);

    NcVar nc_j1yc_re = ncjPFile.addVar("j1yc_re", ncFloat, nc_nX);
    NcVar nc_j1yc_im = ncjPFile.addVar("j1yc_im", ncFloat, nc_nX);

    NcVar nc_j1zc_re = ncjPFile.addVar("j1zc_re", ncFloat, nc_nX);
    NcVar nc_j1zc_im = ncjPFile.addVar("j1zc_im", ncFloat, nc_nX);

    vector<float> JxRe(nXGrid,0);
    vector<float> JxIm(nXGrid,0);
    vector<float> JyRe(nXGrid,0);
    vector<float> JyIm(nXGrid,0);
    vector<float> JzRe(nXGrid,0);
    vector<float> JzIm(nXGrid,0);

    for (int i=0;i<nXGrid;i++) {
       JxRe[i] = j1xc[i].real(); 
       JxIm[i] = j1xc[i].imag(); 
       JyRe[i] = j1yc[i].real(); 
       JyIm[i] = j1yc[i].imag(); 
       JzRe[i] = j1zc[i].real(); 
       JzIm[i] = j1zc[i].imag(); 
    }
    nc_x.putVar(&xGrid[0]);
    nc_j1xc_re.putVar(&JxRe[0]);
    nc_j1xc_im.putVar(&JxIm[0]);
    nc_j1yc_re.putVar(&JyRe[0]);
    nc_j1yc_im.putVar(&JyIm[0]);
    nc_j1zc_re.putVar(&JzRe[0]);
    nc_j1zc_im.putVar(&JzIm[0]);

    cout << "DONE" << endl;

#if CLOCK >= 1
#if not defined(_OPENMP)
        clock_t endTimeFunctor = clock();
        double timeInSecondsFunctor = (endTimeFunctor - startTimeFunctor) / (double)CLOCKS_PER_SEC;
        cout << "Time for this spatial point: " << timeInSecondsFunctor << endl;
        cout << "Time per particle: " << timeInSecondsFunctor / nWork << endl;
#endif
#endif

cout << "Continuing with non functor approach ..." << endl;

#pragma omp parallel for private(istat, tid, spoken)
    for (int iX = 0; iX < nXGrid; iX++) {

#if defined(_OPENMP)
        nThreads = omp_get_num_threads();
        tid = omp_get_thread_num();
        if (tid == 0 && spoken == 0) {
            cout << "tid : " << tid << endl;
            cout << "OMP_NUM_THREADS: " << nThreads << endl;
            spoken = 1;
        }
#endif
        vector<CParticle> ThisParticleList(
            create_particles(xGrid[iX], amu, Z, T_keV[iX], density_m3[iX], nPx, nPy,
                nPz, nThermal, dv, r, b0_CYL));

#if CLOCK >= 1
        clock_t startTime = clock();
#endif
        j1xc[iX] = complex<float>(0, 0);
        j1yc[iX] = complex<float>(0, 0);
        j1zc[iX] = complex<float>(0, 0);

#if LOWMEM_USEPAPI >= 1
        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
#endif

#if F1_WRITE >= 1
        int f1_write_iX = 75;
        ofstream f1File;
        if (iX == f1_write_iX) {
            f1File.open("output/f1.txt", ios::out | ios::trunc);
            f1File << " vx  vy  vz  re(f1) im(f1) " << endl;
        }
#endif

        vector<float> f1(nP);
        vector<complex<float> > f1c(nP);

        for (int iP = 0; iP < nP; iP++) {

            vector<C3Vec> thisOrbitE1_re_XYZ(nSteps, C3Vec(0, 0, 0));
            vector<C3Vec> thisOrbitE1_im_XYZ(nSteps, C3Vec(0, 0, 0));

            vector<C3Vec> thisOrbitB1_re_XYZ(nSteps, C3Vec(0, 0, 0));
            vector<C3Vec> thisOrbitB1_im_XYZ(nSteps, C3Vec(0, 0, 0));

            CParticle thisParticle_XYZ(ThisParticleList[iP]);

            float qOverm = thisParticle_XYZ.q / thisParticle_XYZ.m;

            float Ze = thisParticle_XYZ.q;
#if LOWMEM_ORBIT_WRITE >= 1
            ofstream OrbitFile;
            ofstream v1File;
            ofstream e1_dot_grad_File;
            ofstream df0dv_File;

            int write_iX = 75;
            int write_iP = 33;
            if (iX == write_iX && iP == write_iP) {
                cout << "Write Particle Properties:" << endl;
                cout << " vTh: " << thisParticle_XYZ.vTh << endl;
                cout << " v1: " << thisParticle_XYZ.v_c1 << endl;
                cout << " v2: " << thisParticle_XYZ.v_c2 << endl;
                cout << " v3: " << thisParticle_XYZ.v_c3 << endl;

                OrbitFile.open("output/orbit.txt", ios::out | ios::trunc);
                OrbitFile << "wc / wrf: " << wrf_wc[iX] << endl;
                OrbitFile << " t  x  y  z  re(e1)  im(e1)  re(e2)  im(e2)  re(e3)  "
                             "im(e3)  re(b1)  im(b1)  re(b2)  im(b2)  re(b3)  im(b3) "
                             "status"
                          << endl;
                v1File.open("output/orbit_v1.txt", ios::out | ios::trunc);
                v1File << " t  re(v11)  im(v11)  re(v12)  im(v12)  re(v13)  im(v13)"
                       << endl;
                e1_dot_grad_File.open("output/orbit_e1_dot_grad_df0_dv.txt",
                    ios::out | ios::trunc);
                e1_dot_grad_File << " t  re(v1xb01)  im(v1xb01)  re(v1xb02)  "
                                    "im(v1xb02)  re(v1xb03)  im(v1xb03)"
                                 << endl;
                df0dv_File.open("output/df0dv.txt", ios::out | ios::trunc);
                df0dv_File << " t  vx  vy  vz  valp  vbet  vpar  vper  gyroAngle  "
                              "df0dv_x  df0dv_y  df0dv_z"
                           << endl;
            }
#endif
            // generate orbit and get time-harmonic e along it

            vector<C3Vec> thisOrbit_XYZ(nSteps);
            vector<C3VecI> thisE1c_XYZ(nSteps, C3VecI());
            vector<C3VecI> thisB1c_XYZ(nSteps, C3VecI());
            C3VecI thisV1c_(0, 0, 0), dVc(0, 0, 0), crossTerm(0, 0, 0);
            vector<complex<float> > this_e1_dot_gradvf0(nSteps);
            vector<C3VecI> this_vCrossB1(nSteps);

            for (int i = 0; i < nSteps; i++) {
#if DEBUG_MOVE >= 1
                cout << "Position Before Move: " << thisParticle_XYZ.c1 << "  "
                     << thisParticle_XYZ.c2 << "  " << thisParticle_XYZ.c3 << endl;
                cout << "p.status: " << thisParticle_XYZ.status << endl;
#endif
                thisOrbit_XYZ[i] = C3Vec(thisParticle_XYZ.c1, thisParticle_XYZ.c2,
                    thisParticle_XYZ.c3);
#if GC_ORBITS >= 1
                int MoveStatus = rk4_move_gc(thisParticle_XYZ, dtMin, thisT[i], r, b0_CYL, r_gc,
                    curv_CYL, grad_CYL, bDotGradB, wrf);
#else
                int MoveStatus = rk4_move(thisParticle_XYZ, dtMin, r, b0_CYL);
#endif
                int OverallStatus = max(thisParticle_XYZ.status, MoveStatus);
#if DEBUG_MOVE >= 1
                if (MoveStatus > 0) {
                    cout << "Position After Move: " << thisParticle_XYZ.c1 << "  "
                         << thisParticle_XYZ.c2 << "  " << thisParticle_XYZ.c3 << endl;
                    cout << "ERROR: rk4_move* threw an error" << endl;
                    cout << "MoveStatus: " << MoveStatus << endl;
                    exit(1);
                }
#endif

                C3Vec thisPos(thisParticle_XYZ.c1, thisParticle_XYZ.c2,
                    thisParticle_XYZ.c3);
                C3Vec thisVel_XYZ(thisParticle_XYZ.v_c1, thisParticle_XYZ.v_c2,
                    thisParticle_XYZ.v_c3);
                C3Vec thisB0 = kj_interp1D(thisOrbit_XYZ[i].c1, r, b0_CYL, istat);
#if GC_ORBITS >= 1
                thisVel_XYZ = thisB0 / mag(thisB0) * thisParticle_XYZ.vPar; // vPar vector in XYZ
                cout << thisParticle_XYZ.vPar << "  " << thisParticle_XYZ.vPer << endl;
                kj_print(thisVel_XYZ, "thisVel_XYZ");
                C3Vec gradv_f0_XYZ = maxwellian_df0_dv(thisVel_XYZ, T_keV[iX], density_m3[iX],
                    thisParticle_XYZ.amu, thisParticle_XYZ.Z);
#else
                C3Vec gradv_f0_XYZ = maxwellian_df0_dv(thisVel_XYZ, T_keV[iX], density_m3[iX],
                    thisParticle_XYZ.amu, thisParticle_XYZ.Z);
#endif

                C3VecI E1_XYZ;
                complex<float> _i(0, 1);
                // why is this exp(-iwt) here? surely it's not required for the freq domain calc?
                //E1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ(thisParticle_XYZ, r, e1_CYL, nPhi);
                E1_XYZ = hanningWeight[i] * getE1orB1_XYZ(thisParticle_XYZ, r, e1_CYL, nPhi);
                thisE1c_XYZ[i] = E1_XYZ * (1 - thisParticle_XYZ.status);

                C3VecI B1_XYZ;
                //B1_XYZ = hanningWeight[i] * exp(-_i * wrf * thisT[i]) * getE1orB1_XYZ(thisParticle_XYZ, r, b1_CYL, nPhi);
                B1_XYZ = hanningWeight[i] * getE1orB1_XYZ(thisParticle_XYZ, r, b1_CYL, nPhi);
                thisB1c_XYZ[i] = B1_XYZ * (1 - thisParticle_XYZ.status);

#if DEBUG_MOVE >= 2
                cout << "thisE1c[i].c1: " << thisE1c_XYZ[i].c1 << endl;
                cout << "thisE1c[i].c2: " << thisE1c_XYZ[i].c2 << endl;
                cout << "thisE1c[i].c3: " << thisE1c_XYZ[i].c3 << endl;

                cout << "thisB1c[i].c1: " << thisB1c_XYZ[i].c1 << endl;
                cout << "thisB1c[i].c2: " << thisB1c_XYZ[i].c2 << endl;
                cout << "thisB1c[i].c3: " << thisB1c_XYZ[i].c3 << endl;
#endif
#if DEBUG_FORCE_TERM >= 1
                cout << "thisE1c[i].c1: " << thisE1c_XYZ[i].c1 << endl;
                cout << "thisE1c[i].c2: " << thisE1c_XYZ[i].c2 << endl;
                cout << "thisE1c[i].c3: " << thisE1c_XYZ[i].c3 << endl;

                cout << "thisB1c[i].c1: " << thisB1c_XYZ[i].c1 << endl;
                cout << "thisB1c[i].c2: " << thisB1c_XYZ[i].c2 << endl;
                cout << "thisB1c[i].c3: " << thisB1c_XYZ[i].c3 << endl;

                cout << "thisVel_XYZ.c1: " << thisVel_XYZ.c1 << endl;
                cout << "thisVel_XYZ.c2: " << thisVel_XYZ.c2 << endl;
                cout << "thisVel_XYZ.c3: " << thisVel_XYZ.c3 << endl;

#endif

#if GC_ORBITS >= 1
                // For GC orbits (electrons) use only the orbit parallel piece,
                // since the perp peice will cancel due to E not varying within
                // a cyclotron period.
                //
                // NO - WE DO NOT HAVE thisVel_XYZ for GC!!!!!
                C3Vec orbitParallelUnitVector_XYZ = thisB0 / mag(thisB0);
                // kj_print(orbitParallelUnitVector_XYZ,"unit");
                complex<float> orbitParallel_E = dot(thisE1c_XYZ[i], orbitParallelUnitVector_XYZ);
                float orbitParallel_gradv_f0 = dot(gradv_f0_XYZ, orbitParallelUnitVector_XYZ);
                // cout<<"E : "<<orbitParallel_E<<" gf0:
                // "<<orbitParallel_gradv_f0<<endl;
                this_e1_dot_gradvf0[i] = orbitParallel_E * orbitParallel_gradv_f0;
                // cout<<this_e1_dot_gradvf0[i]<<" e1dotgrad"<<endl;
                complex<float> _full = dot(thisE1c_XYZ[i], gradv_f0_XYZ);
                // cout<<"_full : "<<_full<<endl;
#else
                this_vCrossB1[i] = cross(thisVel_XYZ, thisB1c_XYZ[i]);
                C3VecI this_force = this_vCrossB1[i] + thisE1c_XYZ[i];

                // C3VecI this_force_CYL;
                // float this_t =
                // sqrt(pow(thisParticle_XYZ.c1,2)+pow(thisParticle_XYZ.c2,2));
                // this_force_CYL = rot_CYL_to_XYZ ( this_t, this_force, -1);
                // this_force_CYL.c1 = 0;
                // this_force_CYL.c3 = 0;
                // this_force = rot_CYL_to_XYZ ( this_t, this_force_CYL, +1);

                this_e1_dot_gradvf0[i] = dot(this_force, gradv_f0_XYZ);

                // C3Vec  this_gradv_f0_CYL;
                // this_force_CYL = rot_CYL_to_XYZ ( this_t, this_force, -1);
                // this_gradv_f0_CYL = rot_CYL_to_XYZ ( this_t, gradv_f0_XYZ, -1);
                // this_e1_dot_gradvf0[i] = dot(this_force_CYL, this_gradv_f0_CYL);
#endif

#if LOWMEM_ORBIT_WRITE >= 1
                if (iX == write_iX && iP == write_iP) {
                    df0dv_File << scientific;
                    df0dv_File << thisT[i] << "    " << thisVel_XYZ.c1 << "    "
                               << thisVel_XYZ.c2 << "    " << thisVel_XYZ.c3 << "    "
                               << thisParticle_XYZ.vAlp << "    " << thisParticle_XYZ.vBet
                               << "    " << thisParticle_XYZ.vPar << "    "
                               << thisParticle_XYZ.vPer << "    " << thisParticle_XYZ.phs
                               << "    " << gradv_f0_XYZ.c1 << "    " << gradv_f0_XYZ.c2
                               << "    " << gradv_f0_XYZ.c3 << endl;
                }

                if (iX == write_iX && iP == write_iP) {
                    OrbitFile << scientific;
                    OrbitFile << thisT[i] << "    " << thisPos.c1 << "    " << thisPos.c2
                              << "    " << thisPos.c3 << "    " << real(thisE1c_XYZ[i].c1)
                              << "    " << imag(thisE1c_XYZ[i].c1) << "    "
                              << real(thisE1c_XYZ[i].c2) << "    "
                              << imag(thisE1c_XYZ[i].c2) << "    "
                              << real(thisE1c_XYZ[i].c3) << "    "
                              << imag(thisE1c_XYZ[i].c3) << "    "
                              << real(thisB1c_XYZ[i].c1) << "    "
                              << imag(thisB1c_XYZ[i].c1) << "    "
                              << real(thisB1c_XYZ[i].c2) << "    "
                              << imag(thisB1c_XYZ[i].c2) << "    "
                              << real(thisB1c_XYZ[i].c3) << "    "
                              << imag(thisB1c_XYZ[i].c3) << "    "
                              << real(this_vCrossB1[i].c1) << "    "
                              << imag(this_vCrossB1[i].c1) << "    "
                              << real(this_vCrossB1[i].c2) << "    "
                              << imag(this_vCrossB1[i].c2) << "    "
                              << real(this_vCrossB1[i].c3) << "    "
                              << imag(this_vCrossB1[i].c3) << "    "
                              << thisParticle_XYZ.status << endl;
                }
                if (iX == write_iX && iP == write_iP) {
                    e1_dot_grad_File << scientific;
                    e1_dot_grad_File << thisT[i] << "    " << real(this_e1_dot_gradvf0[i])
                                     << "    " << imag(this_e1_dot_gradvf0[i]) << endl;
                }
#endif
            }
#if LOWMEM_ORBIT_WRITE >= 1
            if (iX == write_iX && iP == write_iP) {
                OrbitFile.close();
            }
#endif
            complex<float> this_f1c = -qOverm * intVecArray(thisT, this_e1_dot_gradvf0);

#if LOWMEM_ORBIT_WRITE >= 1
            if (iX == write_iX && iP == write_iP) {

                complex<float> tmp = 0.0;
                for (int i = 0; i < nSteps; i++) {
                    tmp += -qOverm * this_e1_dot_gradvf0[i] * dtMin;
                    v1File << thisT[i] << "    " << real(tmp) << "    " << imag(tmp)
                           << endl;
                }
            }
#endif
            f1c[iP] = -this_f1c;

            float v0x_i = ThisParticleList[iP].v_c1;
            float v0y_i = ThisParticleList[iP].v_c2;
            float v0z_i = ThisParticleList[iP].v_c3;

            float h = dv * Ze;

#pragma omp critical // "atomic" does not work for complex numbers
            {
                j1xc[iX] += h * (v0x_i * f1c[iP]);
                j1yc[iX] += h * (v0y_i * f1c[iP]);
                j1zc[iX] += h * (v0z_i * f1c[iP]);
            }

#if F1_WRITE >= 1
            if (iX == f1_write_iX) {
                f1File << scientific;
                f1File << showpos;
                f1File << v0x_i << "    " << v0y_i << "    " << v0z_i << "    "
                       << real(f1c[iP]) << "    " << imag(f1c[iP]) << endl;
            }
#endif
        }

#if CLOCK >= 1
#if not defined(_OPENMP)
        clock_t endTime = clock();
        double timeInSeconds = (endTime - startTime) / (double)CLOCKS_PER_SEC;
        cout << "Time for this spatial point: " << timeInSeconds << endl;
        cout << "Time per particle: " << timeInSeconds / nP << endl;
#endif
#endif

#if LOWMEM_USEPAPI >= 1
        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
        printf("\nLOWMEM Oribit calculation performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif

#if USEPAPI >= 1
        printf("\nGet e(t) and integrate performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            eT_realTime, eT_cpuTime, eT_flpIns, eT_mFlops / (nJp - 1));
        printf("\nGet v(t) and integrate performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            vT_realTime, vT_cpuTime, vT_flpIns, vT_mFlops / (nJp - 1));

        cpuTime0 = cpuTime;
        realTime0 = realTime;
        flpIns0 = flpIns;
        papiReturn = PAPI_flops(&realTime, &cpuTime, &flpIns, &mFlops);
        printf("\nj(t) performance ...\n");
        printf(
            "Real_time:\t%f\nProc_time:\t%f\nTotal flpins:\t%lld\nMFLOPS:\t\t%f\n",
            realTime - realTime0, cpuTime - cpuTime0, flpIns - flpIns0, mFlops);
#endif

#if __SAVE_ORBITS__ >= 1
        // Write orbits to file

        cout << "Writing orbits to file ... " << endl;

        stringstream ncOrbitsFileName;
        ncOrbitsFileName << "output/orbits_";
        ncOrbitsFileName << setw(3) << setfill('0') << iX;
        ncOrbitsFileName << ".nc";

        try {
            // Really need to fix this but I don't know how to
            // write a vector of structures using netCDF yet.

            NcFile ncOrbitsFile(ncOrbitsFileName.str().c_str(), NcFile::replace);

            NcDim nc_nP = ncOrbitsFile.addDim("nP", this_particles_XYZ.size());
            NcDim nc_nSteps = ncOrbitsFile.addDim("nSteps", nSteps);
            NcDim nc_nJp = ncOrbitsFile.addDim("nJp", nJp);

            vector<NcDim> nc_nPxnSteps(2);
            nc_nPxnSteps[0] = nc_nP;
            nc_nPxnSteps[1] = nc_nSteps;

            vector<NcDim> nc_nPxnJpxnSteps(3);
            nc_nPxnJpxnSteps[0] = nc_nP;
            nc_nPxnJpxnSteps[1] = nc_nJp;
            nc_nPxnJpxnSteps[2] = nc_nSteps;

            NcVar nc_t = ncOrbitsFile.addVar("t", ncFloat, nc_nSteps);

            NcVar nc_x = ncOrbitsFile.addVar("x", ncFloat, nc_nPxnSteps);
            NcVar nc_y = ncOrbitsFile.addVar("y", ncFloat, nc_nPxnSteps);
            NcVar nc_z = ncOrbitsFile.addVar("z", ncFloat, nc_nPxnSteps);

            NcVar nc_vx = ncOrbitsFile.addVar("vx", ncFloat, nc_nPxnSteps);
            NcVar nc_vy = ncOrbitsFile.addVar("vy", ncFloat, nc_nPxnSteps);
            NcVar nc_vz = ncOrbitsFile.addVar("vz", ncFloat, nc_nPxnSteps);

            NcVar nc_e1_x = ncOrbitsFile.addVar("e1_x", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_y = ncOrbitsFile.addVar("e1_y", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_z = ncOrbitsFile.addVar("e1_z", ncFloat, nc_nPxnSteps);

            NcVar nc_e1_x_re = ncOrbitsFile.addVar("e1_x_re", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_y_re = ncOrbitsFile.addVar("e1_y_re", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_z_re = ncOrbitsFile.addVar("e1_z_re", ncFloat, nc_nPxnSteps);

            NcVar nc_e1_x_im = ncOrbitsFile.addVar("e1_x_im", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_y_im = ncOrbitsFile.addVar("e1_y_im", ncFloat, nc_nPxnSteps);
            NcVar nc_e1_z_im = ncOrbitsFile.addVar("e1_z_im", ncFloat, nc_nPxnSteps);

            NcVar nc_v1_x = ncOrbitsFile.addVar("v1x", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_y = ncOrbitsFile.addVar("v1y", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_z = ncOrbitsFile.addVar("v1z", ncFloat, nc_nPxnJpxnSteps);

            NcVar nc_v1_x_re = ncOrbitsFile.addVar("v1x_re", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_y_re = ncOrbitsFile.addVar("v1y_re", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_z_re = ncOrbitsFile.addVar("v1z_re", ncFloat, nc_nPxnJpxnSteps);

            NcVar nc_v1_x_im = ncOrbitsFile.addVar("v1x_im", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_y_im = ncOrbitsFile.addVar("v1y_im", ncFloat, nc_nPxnJpxnSteps);
            NcVar nc_v1_z_im = ncOrbitsFile.addVar("v1z_im", ncFloat, nc_nPxnJpxnSteps);

            vector<size_t> startpA(2);
            vector<size_t> countpA(2);
            for (int iP = 0; iP < this_particles_XYZ.size(); iP++) {

                startpA[0] = iP;
                startpA[1] = 0;
                countpA[0] = 1;
                countpA[1] = nSteps;

                vector<float> tmpData(nSteps, 0);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_XYZ[iP][iS].c1;
                }
                nc_x.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_XYZ[iP][iS].c2;
                }
                nc_y.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_XYZ[iP][iS].c3;
                }
                nc_z.putVar(startpA, countpA, &tmpData[0]);

                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_v_XYZ[iP][iS].c1;
                }
                nc_vx.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_v_XYZ[iP][iS].c2;
                }
                nc_vy.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = orbits_v_XYZ[iP][iS].c3;
                }
                nc_vz.putVar(startpA, countpA, &tmpData[0]);

                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = e1[iP][iS].c1;
                }
                nc_e1_x.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = e1[iP][iS].c2;
                }
                nc_e1_y.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = e1[iP][iS].c3;
                }
                nc_e1_z.putVar(startpA, countpA, &tmpData[0]);

                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = real(e1c[iP][iS].c1);
                }
                nc_e1_x_re.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = real(e1c[iP][iS].c2);
                }
                nc_e1_y_re.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = real(e1c[iP][iS].c3);
                }
                nc_e1_z_re.putVar(startpA, countpA, &tmpData[0]);

                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = imag(e1c[iP][iS].c1);
                }
                nc_e1_x_im.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = imag(e1c[iP][iS].c2);
                }
                nc_e1_y_im.putVar(startpA, countpA, &tmpData[0]);
                for (int iS = 0; iS < nSteps; iS++) {
                    tmpData[iS] = imag(e1c[iP][iS].c3);
                }
                nc_e1_z_im.putVar(startpA, countpA, &tmpData[0]);
            }

            vector<size_t> startpB(3);
            vector<size_t> countpB(3);
            for (int iP = 0; iP < this_particles_XYZ.size(); iP++) {
                for (int iJ = 0; iJ < nJp; iJ++) {

                    startpB[0] = iP;
                    startpB[1] = iJ;
                    startpB[2] = 0;
                    countpB[0] = 1;
                    countpB[1] = 1;
                    countpB[2] = nSteps;

                    vector<float> tmpData(nSteps, 0);

                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = v1[iP][iJ][iS].c1;
                    }
                    nc_v1_x.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = v1[iP][iJ][iS].c2;
                    }
                    nc_v1_y.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = v1[iP][iJ][iS].c3;
                    }
                    nc_v1_z.putVar(startpB, countpB, &tmpData[0]);

                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = real(v1c[iP][iJ][iS].c1);
                    }
                    nc_v1_x_re.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = real(v1c[iP][iJ][iS].c2);
                    }
                    nc_v1_y_re.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = real(v1c[iP][iJ][iS].c3);
                    }
                    nc_v1_z_re.putVar(startpB, countpB, &tmpData[0]);

                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = imag(v1c[iP][iJ][iS].c1);
                    }
                    nc_v1_x_im.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = imag(v1c[iP][iJ][iS].c2);
                    }
                    nc_v1_y_im.putVar(startpB, countpB, &tmpData[0]);
                    for (int iS = 0; iS < nSteps; iS++) {
                        tmpData[iS] = imag(v1c[iP][iJ][iS].c3);
                    }
                    nc_v1_z_im.putVar(startpB, countpB, &tmpData[0]);
                }
            }

            vector<size_t> startp(1, 0);
            vector<size_t> countp(1, nSteps);

            nc_t.putVar(startp, countp, &thisT[0]);

        } catch (exceptions::NcException& e) {
            cout << "NetCDF: unknown error" << endl;
            e.what();
            exit(1);
        }

// cout << "DONE" << endl;
#endif

    } // End of xGrid loop

    // Write current(s) to file

    // cout << "Writing jP to file ... ";

    for (int iX = 0; iX < nXGrid; iX++) {

        stringstream ncjPFileName;
        ncjPFileName << "output/";
        // check directory exists
        struct stat st;
        if (stat(ncjPFileName.str().c_str(), &st) != 1) {
            int mkDirStat = mkdir(ncjPFileName.str().c_str(),
                S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
        }
        ncjPFileName << "/jP_";
        ncjPFileName << setw(3) << setfill('0') << iX;
        ncjPFileName << ".nc";
#if DEBUGLEVEL >= 1
        cout << ncjPFileName.str().c_str() << endl;
#endif

        NcFile ncjPFile(ncjPFileName.str().c_str(), NcFile::replace);

        NcDim nc_nJp = ncjPFile.addDim("nJp", nJp);
        NcDim nc_scalar = ncjPFile.addDim("scalar", 1);

        NcVar nc_t = ncjPFile.addVar("t", ncFloat, nc_nJp);

        NcVar nc_x = ncjPFile.addVar("x", ncFloat, nc_scalar);
        NcVar nc_freq = ncjPFile.addVar("freq", ncFloat, nc_scalar);

        NcVar nc_j1x = ncjPFile.addVar("j1x", ncFloat, nc_nJp);
        NcVar nc_j1y = ncjPFile.addVar("j1y", ncFloat, nc_nJp);
        NcVar nc_j1z = ncjPFile.addVar("j1z", ncFloat, nc_nJp);

        NcVar nc_j1xc_re = ncjPFile.addVar("j1xc_re", ncFloat, nc_scalar);
        NcVar nc_j1xc_im = ncjPFile.addVar("j1xc_im", ncFloat, nc_scalar);

        NcVar nc_j1yc_re = ncjPFile.addVar("j1yc_re", ncFloat, nc_scalar);
        NcVar nc_j1yc_im = ncjPFile.addVar("j1yc_im", ncFloat, nc_scalar);

        NcVar nc_j1zc_re = ncjPFile.addVar("j1zc_re", ncFloat, nc_scalar);
        NcVar nc_j1zc_im = ncjPFile.addVar("j1zc_im", ncFloat, nc_scalar);

        nc_x.putVar(&xGrid[iX]);
        nc_freq.putVar(&freq);

        vector<size_t> startp(1, 0);
        vector<size_t> countp(1, nJp);

        float tmpJxRe = real(j1xc[iX]);
        float tmpJxIm = imag(j1xc[iX]);
        nc_j1xc_re.putVar(&tmpJxRe);
        nc_j1xc_im.putVar(&tmpJxIm);

        float tmpJyRe = real(j1yc[iX]);
        float tmpJyIm = imag(j1yc[iX]);
        nc_j1yc_re.putVar(&tmpJyRe);
        nc_j1yc_im.putVar(&tmpJyIm);

        float tmpJzRe = real(j1zc[iX]);
        float tmpJzIm = imag(j1zc[iX]);
        nc_j1zc_re.putVar(&tmpJzRe);
        nc_j1zc_im.putVar(&tmpJzIm);
    }

    // ProfilerStop();

    cout << "DONE" << endl;

#if CLOCK >= 1
    clock_t ProgramTime_ = clock();
    double ProgramTimeInSeconds = (ProgramTime_ - ProgramTime) / (double)CLOCKS_PER_SEC;
#if defined(_OPENMP)
    ProgramTimeInSeconds = ProgramTimeInSeconds / nThreads;
    cout << "nThreads: " << nThreads << endl;
#endif
    cout << "Total Time [s]: " << ProgramTimeInSeconds << endl;
    cout << "Total Time [m]: " << ProgramTimeInSeconds / 60.0 << endl;
    cout << "Total Time [h]: " << ProgramTimeInSeconds / 3600.0 << endl;
#endif
    return EXIT_SUCCESS;
}
