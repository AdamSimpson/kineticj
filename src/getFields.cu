#include "hip/hip_runtime.h"
#include "getFields.hpp"

#ifdef __HIPCC__
HOST DEVICE
C3<thrust::complex<float> > getE1orB1_XYZ(CParticle& p_XYZ, float *rVec, C3<thrust::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<thrust::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    thrust::complex<float> ii(0, 1);

    E1_XYZ = thrust::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}
#endif

HOST
C3<std::complex<float> > getE1orB1_XYZ(CParticle& p_XYZ, float *rVec, C3<std::complex<float> > *E1Vec_CYL, int nR, int nPhi)
{

    float _r = sqrt(pow(p_XYZ.c1, 2) + pow(p_XYZ.c2, 2));
    float _p = atan2(p_XYZ.c2, p_XYZ.c1);

    C3<std::complex<float> > E1_CYL, E1_XYZ;

    int status = 0;
    E1_CYL = kj_interp1D(_r, rVec, E1Vec_CYL, nR, status);
    p_XYZ.status = max(p_XYZ.status, status);

    std::complex<float> ii(0, 1);

    E1_XYZ = std::exp(ii * float(nPhi * _p)) * rot_CYL_to_XYZ(_p, E1_CYL, 1);

    return E1_XYZ;
}


